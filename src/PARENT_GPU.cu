
#include <hip/hip_runtime.h>
#include <iostream>

#define PRECISION double

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t return_code, const char *file, int line)
{
   if (return_code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(return_code), file, line);
      exit(return_code);
   }
}

using namespace std;

class GPU_RAM_Block{
	public:
		unsigned char type;
		unsigned int first_dof;
		unsigned int last_dof;
		unsigned int n_dofs;
		PRECISION *cpu_ram_start;
		PRECISION *cpu_ram_end; 
		PRECISION *gpu_ram_start;
	       	PRECISION *gpu_ram_end;
		unsigned long long int n_bytes;

	GPU_RAM_Block(unsigned char type, unsigned int first_dof, unsigned int n_dofs, 
			PRECISION *cpu_ram_start, PRECISION *gpu_ram_start, unsigned long long int n_bytes){
		
		this->type = type;
		this->first_dof = first_dof;
		this->n_dofs = n_dofs;
		this->last_dof = first_dof + n_dofs - 1;
		this->cpu_ram_start = cpu_ram_start;
		this->cpu_ram_end = cpu_ram_start + (n_bytes - 1) / sizeof(PRECISION);
		this->gpu_ram_start = gpu_ram_start;
		this->gpu_ram_end = gpu_ram_start + (n_bytes - 1) / sizeof(PRECISION);
		this->n_bytes = n_bytes;

	}


	void load_GPU(){
		gpuErrchk(hipMemcpy(cpu_ram_start, gpu_ram_start, n_bytes, hipMemcpyHostToDevice));
	}	



};








int main(){


return 0;
}



