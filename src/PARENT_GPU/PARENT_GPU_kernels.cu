#include "hip/hip_runtime.h"
// #define FAST_MATH
#ifdef FAST_MATH
    #define COS __cosf
    #define SIN __sinf
    #define EXP __expf
    #define LOG __logf
#else 
    #define COS cos
    #define SIN sin
    #define EXP exp
    #define LOG log
#endif


__global__ void histo2D(PRECISION* __restrict__ traj1, PRECISION* __restrict__ traj2, const int numFrames,
                        unsigned int* __restrict__ histo, const int n_bins,
                        PRECISION binSize1, PRECISION binSize2, PRECISION min1,
                        PRECISION min2) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
    
  while (idx < numFrames) {
    atomicAdd(&histo[int((traj1[idx] - min1) / binSize1) * n_bins +
                     int((traj2[idx] - min2) / binSize2)],
              1);
    idx += offset;
  }
}

__global__ void histo2D_shared_block(PRECISION* __restrict__ traj1, PRECISION* __restrict__ traj2, const int numFrames,
                        unsigned int* __restrict__ histo, const int n_bins,
                        PRECISION binSize1, PRECISION binSize2, PRECISION min1,
                        PRECISION min2) {
                        
    extern __shared__ unsigned int histo_block[];

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
    int tid = threadIdx.x;
    int n_bins_total = n_bins * n_bins;
                        
    while(tid < n_bins_total){
        histo_block[tid] = 0;
        tid += blockDim.x;
    }
    
  while (idx < numFrames) {
    atomicAdd(&histo_block[int((traj1[idx] - min1) / binSize1) * n_bins +
                     int((traj2[idx] - min2) / binSize2)],
              1);
    idx += offset;
  }
  
  
  __syncthreads();
  
  tid = threadIdx.x;
  while(tid < n_bins_total){
    atomicAdd(&(histo[tid]), histo_block[tid]);
    tid += blockDim.x;
  }
  
}

__global__ void cu_baEnt(unsigned int *histo, const int numFrames,
                         const int bins1, const int bins2, PRECISION binSize1,
                         PRECISION binSize2, PRECISION min1, PRECISION min2,
                         PRECISION *plnpsum, unsigned int *occupbins) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
  while (idx < bins1 * bins2) {
    PRECISION blen = min1 + binSize1 / 2.0 + binSize1 * (idx / bins2);
    PRECISION theta = min2 + binSize2 / 2.0 + binSize2 * (idx % bins2);
    PRECISION probDens = histo[idx] / (numFrames * binSize1 * binSize2 * blen *
                                       blen * SIN(theta));
    if (probDens > 0) {
      atomicAdd(plnpsum, blen * blen * SIN(theta) * probDens * LOG(probDens) *
                             binSize1 * binSize2);
      atomicAdd(occupbins, 1);
    }
    idx += offset;
  }
}

__global__ void cu_bdEnt(unsigned int *histo, const int numFrames,
                         const int bins1, const int bins2, PRECISION binSize1,
                         PRECISION binSize2, PRECISION min1, PRECISION *plnpsum,
                         unsigned int *occupbins) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
  while (idx < bins1 * bins2) {
    PRECISION blen = min1 + binSize1 / 2.0 + binSize1 * (idx / bins2);
    PRECISION probDens =
        histo[idx] / (numFrames * binSize1 * binSize2 * blen * blen);
    if (probDens > 0) {
      atomicAdd(plnpsum,
                blen * blen * probDens * LOG(probDens) * binSize1 * binSize2);
      atomicAdd(occupbins, 1);
    }
    idx += offset;
  }
}

__global__ void cu_adEnt(unsigned int *histo, const int numFrames,
                         const int bins1, const int bins2, PRECISION binSize1,
                         PRECISION binSize2, PRECISION min1, PRECISION *plnpsum,
                         unsigned int *occupbins) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
  while (idx < bins1 * bins2) {
    PRECISION theta = min1 + binSize1 / 2.0 + binSize1 * (idx / bins2);
    PRECISION probDens =
        histo[idx] / (numFrames * binSize1 * binSize2 * SIN(theta));
    if (probDens > 0) {
      atomicAdd(plnpsum,
                SIN(theta) * probDens * LOG(probDens) * binSize1 * binSize2);
      atomicAdd(occupbins, 1);
    }
    idx += offset;
  }
}

__global__ void cu_bbEnt(unsigned int *histo, const int numFrames,
                         const int bins, PRECISION binSize1, PRECISION binSize2,
                         PRECISION min1, PRECISION min2, PRECISION *plnpsum,
                         unsigned int *occupbins) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
  while (idx < bins * bins) {
    PRECISION blen1 = min1 + binSize1 / 2.0 + binSize1 * (idx / bins);
    PRECISION blen2 = min2 + binSize2 / 2.0 + binSize2 * (idx % bins);
    PRECISION probDens = histo[idx] / (numFrames * binSize1 * binSize2 * blen1 *
                                       blen1 * blen2 * blen2);
    if (probDens > 0) {
      atomicAdd(plnpsum, blen1 * blen1 * blen2 * blen2 * probDens *
                             LOG(probDens) * binSize1 * binSize2);
      atomicAdd(occupbins, 1);
    }
    idx += offset;
  }
}

__global__ void cu_aaEnt(unsigned int *histo, const int numFrames,
                         const int bins, PRECISION binSize1, PRECISION binSize2,
                         PRECISION min1, PRECISION min2, PRECISION *plnpsum,
                         unsigned int *occupbins) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
  while (idx < bins * bins) {
    PRECISION theta1 = min1 + binSize1 / 2.0 + binSize1 * (idx / bins);
    PRECISION theta2 = min2 + binSize2 / 2.0 + binSize2 * (idx % bins);
    PRECISION probDens = histo[idx] / (numFrames * binSize1 * binSize2 *
                                       SIN(theta1) * SIN(theta2));
    if (probDens > 0) {
      atomicAdd(plnpsum, SIN(theta1) * SIN(theta2) * probDens * LOG(probDens) *
                             binSize1 * binSize2);
      atomicAdd(occupbins, 1);
    }
    idx += offset;
  }
}

__global__ void cu_ddEnt(unsigned int *histo, const int numFrames,
                         const int bins, PRECISION binSize1, PRECISION binSize2,
                         PRECISION *plnpsum, unsigned int *occupbins) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
  while (idx < bins * bins) {
    PRECISION probDens = histo[idx] / (numFrames * binSize1 * binSize2);
    if (probDens > 0) {
      atomicAdd(plnpsum, probDens * LOG(probDens) * binSize1 * binSize2);
      atomicAdd(occupbins, 1);
    }
    idx += offset;
  }
}


