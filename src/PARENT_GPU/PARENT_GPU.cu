#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>
#include <sys/time.h>

#include "../util/io/io.h"
#include "../util/types.h"

#define PRECISION double
#define MODFITNBINS 100
#define WARPMULTIPLES 1
#define MEMORY_USAGE 0.8 //GPU
#define RAM_USAGE 0.8 //CPU
#define DEBUG false

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t return_code, const char *file, int line)
{
   if (return_code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(return_code), file, line);
      exit(return_code);
   }
}

using namespace std;



struct Bins {
	unsigned int bonds1D;
	unsigned int angles1D;
	unsigned int dihedrals1D;
	unsigned int bonds2D;
	unsigned int angles2D;
	unsigned int dihedrals2D;
};


unsigned char get_dof_type_from_id(unsigned int dof_id, unsigned int n_dihedrals)
{
	if(dof_id < n_dihedrals + 2) return TYPE_B;
	if(dof_id < 2 * n_dihedrals + 3) return TYPE_A;
	return TYPE_D;
}



unsigned int get_min_id_for_type(unsigned char type, unsigned int n_dihedrals)
{
	switch(type) {
  		case TYPE_B:
    			return 0;
  		case TYPE_A:
    			return n_dihedrals + 2;
		case TYPE_D:
    			return 2 * n_dihedrals + 3;
	}
	return 42;
}



unsigned int get_max_id_for_type(unsigned char type, unsigned int n_dihedrals)
{
	switch(type) {
  		case TYPE_B:
    			return n_dihedrals + 1;
  		case TYPE_A:
    			return 2 * n_dihedrals + 2;
		case TYPE_D:
    			return 3 * n_dihedrals + 2;
	}
	return 42;
}




class GPU_RAM_Block
{
	public:
		unsigned char type;
		unsigned int first_dof;
		unsigned int last_dof;
		unsigned int n_dofs;
		unsigned long long int n_bytes;
		PRECISION* cpu_ram_start;

		GPU_RAM_Block(PRECISION* cpu_ram_start, unsigned int first_dof, unsigned int last_dof, unsigned int n_frames, unsigned int n_dihedrals)
		{
			this->cpu_ram_start = cpu_ram_start;
			this->type = get_dof_type_from_id(first_dof, n_dihedrals);
			this->first_dof = first_dof;
			this->last_dof = last_dof;
			n_dofs = last_dof - first_dof + 1;
			n_bytes = n_dofs * n_frames * sizeof(PRECISION);
		}


		void deploy(PRECISION* gpu_ram_start)
		{
			gpuErrchk(hipMemcpy(cpu_ram_start, gpu_ram_start, n_bytes, hipMemcpyHostToDevice));
		}	
};



class CPU_RAM_Block
{
	public:
		unsigned int dof_id_start;
		unsigned int dof_id_end; //inclusive
		unsigned int n_dofs;
		int type_id_start[3] = {-1,-1,-1};
		int type_id_end[3] = {-1,-1,-1}; //inclusive
		unsigned int type_n_dofs[3] = {0,0,0};
		unsigned int gpu_ram_blocks_per_type[3];
		unsigned int n_dihedrals;
		unsigned int gpu_dofs_per_block;
		unsigned int n_frames;
		ifstream* bat_file;
		streamoff file_dofs_begin; 
		unsigned char precision_traj;
		vector<GPU_RAM_Block> blocks;
		PRECISION* block_start;
		PRECISION* minima;
		PRECISION* maxima;
		bool extrema_calculated = false;
		unsigned int n_bins;
		PRECISION* result_entropy1D;
		PRECISION* type_addr[3];
		PRECISION* bonds;
		PRECISION* angles;
		PRECISION* dihedrals;
	
	CPU_RAM_Block(unsigned int dof_id_start, unsigned int dof_id_end, unsigned int gpu_dofs_per_block, unsigned int n_dihedrals, unsigned int n_frames, ifstream* bat_file, 
			streamoff file_dofs_begin, unsigned char precision_traj, PRECISION* minima, PRECISION* maxima, unsigned int n_bins, PRECISION* result_entropy1D)
	{
		cout<<"\nRAMBLOCK "<<dof_id_start<<" "<<dof_id_end<<endl;
		cout<<"DOFS BEFORE "<<type_n_dofs[0]<<" "<<type_n_dofs[1]<<" "<<type_n_dofs[2]<<endl;

		
		this->dof_id_start = dof_id_start;
		this->dof_id_end = dof_id_end;
		this->n_dofs = dof_id_end - dof_id_start + 1;
		this->n_dihedrals = n_dihedrals;
		this->gpu_dofs_per_block = gpu_dofs_per_block;
		this->n_frames = n_frames;
		this->bat_file = bat_file; 
		this->file_dofs_begin = file_dofs_begin;
		this->precision_traj = precision_traj;
		this->minima = minima;
		this->maxima = maxima;  
		this->n_bins = n_bins;		
		this->result_entropy1D = result_entropy1D;
	
		
		for(unsigned short type = get_dof_type_from_id(dof_id_start, n_dihedrals); type <= get_dof_type_from_id(dof_id_end, n_dihedrals); type++){
			cout<<"TYPE "<<type<<endl;
			if(dof_id_start < get_min_id_for_type(type, n_dihedrals)) 
			{
				type_id_start[type] = get_min_id_for_type(type, n_dihedrals); 
			}
			else
			{
				type_id_start[type] = dof_id_start;
			}
			
			if(dof_id_end > get_max_id_for_type(type, n_dihedrals)) 
			{
				type_id_end[type] = get_max_id_for_type(type, n_dihedrals);
			}
			else
			{
				type_id_end[type] = dof_id_end;
			} 
			type_n_dofs[type] = type_id_end[type] - type_id_start[type] + 1;
			
			gpu_ram_blocks_per_type[type] = type_n_dofs[type] / gpu_dofs_per_block;
			if(type_n_dofs[type] % gpu_dofs_per_block > 0) gpu_ram_blocks_per_type[type] += 1;

		}
		cout<<"DOFS AFTER "<<type_n_dofs[0]<<" "<<type_n_dofs[1]<<" "<<type_n_dofs[2]<<endl<<endl;
	}


	void deploy(PRECISION* block_start){
		
		this->block_start = block_start;
		type_addr[TYPE_B] = block_start;
		type_addr[TYPE_A] = block_start + type_n_dofs[TYPE_B] * n_frames;
		type_addr[TYPE_D] = block_start + (type_n_dofs[TYPE_B] + type_n_dofs[TYPE_A]) * n_frames;
		bonds = type_addr[TYPE_B];
		angles = type_addr[TYPE_A];
		dihedrals = type_addr[TYPE_D];

		//cout<<bonds<<" "<<angles<<" "<<dihedrals<<endl;


		load_dofs(block_start);
		modfit_dihedrals();
		if(!extrema_calculated) 
		{
			calculate_extrema();
			calculate_entropy1D();
			extrema_calculated = true;
		}
			
		
		blocks.clear();
		for(unsigned char type = get_dof_type_from_id(dof_id_start, n_dihedrals); type <= get_dof_type_from_id(dof_id_end, n_dihedrals); type++)
		{
			for(unsigned int i = 0; i < gpu_ram_blocks_per_type[type]; i++)
			{
				unsigned int block_id_start = type_id_start[type] + i *  gpu_dofs_per_block;
				unsigned int block_id_end = type_id_start[type] + (i + 1) *  gpu_dofs_per_block - 1; 
				if (block_id_end > type_id_end[type]) block_id_end = type_id_end[type];
				PRECISION* cpu_ram_start = block_start + (block_id_start - dof_id_start) * n_frames;	
				blocks.push_back( *new GPU_RAM_Block(cpu_ram_start, block_id_start, block_id_end, n_frames, n_dihedrals) ); 
			} 
		}		
	}


	unsigned char load_dofs(PRECISION* dof_bank)
	{
		unsigned char fail=0;
		bat_file->seekg(file_dofs_begin);
		for(unsigned int frame=0; frame<n_frames; frame++)
		{
		        //to read a frame from the .bat trajectory
		        double ddummy[6];
			float fdummy[11];
			unsigned int a_start = get_min_id_for_type(TYPE_A, n_dihedrals); 
			unsigned int d_start = get_min_id_for_type(TYPE_D, n_dihedrals); 
			unsigned int b_counter = 0;		       	
			unsigned int a_counter = a_start; 		       	
			unsigned int d_counter = d_start; 
			unsigned int b_counter_local = 0;		       	
			unsigned int a_counter_local = 0; 		       	
			unsigned int d_counter_local = 0;		       			       	
		    
		        if(frame % 100000 == 0) 
			{
		            cout<<"Reading frame "<<frame<<" and the following.\n";   //every 10000 frames issue an information to stdout
		            cout.flush();
		        }
		        
		
		        bat_file->read((char*)fdummy, 11*sizeof(float));//read time, precision and box vectors to dummies
		        fail=fail | (bat_file->rdstate() & std::ifstream::failbit);	
			unsigned char inc;		
		        if(precision_traj==1) 
			{
				inc = sizeof(double);//if trajectory is in double precision
			}
			else
			{
				inc = sizeof(float);
			}	
		        bat_file->read((char*)ddummy, 6 * inc);//external coordinates to dummies
		        fail=fail | (bat_file->rdstate() & std::ifstream::failbit);
		            
		        bat_file->read((char*)ddummy, inc);//read the lengths of the two bonds connecting the root atoms (internal coordinates)
		        fail=fail | (bat_file->rdstate() & std::ifstream::failbit);
		        if( (b_counter >= type_id_start[TYPE_B]) && (b_counter <= type_id_end[TYPE_B]) ) bonds[b_counter_local++ * n_frames + frame] = ddummy[0];
			b_counter++;

			bat_file->read((char*)ddummy, inc);//read the lengths of the two bonds connecting the root atoms (internal coordinates)
		        fail=fail | (bat_file->rdstate() & std::ifstream::failbit);
		        if( (b_counter >= type_id_start[TYPE_B]) && (b_counter <= type_id_end[TYPE_B]) ) bonds[b_counter_local++ * n_frames + frame] = ddummy[0];
			b_counter++;
		            
		        bat_file->read((char*)ddummy, inc);//and the angle between the two rootbonds (internal coordinates)
		        fail=fail | (bat_file->rdstate() & std::ifstream::failbit);
			if( (a_counter >= type_id_start[TYPE_A]) && (a_counter <= type_id_end[TYPE_A]) ) angles[a_counter_local++ * n_frames + frame] = ddummy[0];
			a_counter++;  
		        
			for(int i = 0; i < n_dihedrals; i++) 
		        { //then for all dihedrals in the system
				bat_file->read((char*)ddummy, inc);//read the bondlength between the last two atoms in the dihedral
		                fail=fail | (bat_file->rdstate() & std::ifstream::failbit);		              
		        	if( (b_counter >= type_id_start[TYPE_B]) && (b_counter <= type_id_end[TYPE_B]) ) bonds[b_counter_local++ * n_frames + frame] = ddummy[0];
				b_counter++;

		                bat_file->read((char*)ddummy, inc);//read the angle between the last threee atoms of the dihedral#
		                fail=fail | (bat_file->rdstate() & std::ifstream::failbit);
		        	if( (a_counter >= type_id_start[TYPE_A]) && (a_counter <= type_id_end[TYPE_A]) ) angles[a_counter_local++ * n_frames + frame] = ddummy[0];
				a_counter++;

		                bat_file->read((char*)ddummy, inc);//and the value of the dihedral itself
		                fail=fail | (bat_file->rdstate() & std::ifstream::failbit);
		        	if( (d_counter >= type_id_start[TYPE_D]) && (d_counter <= type_id_end[TYPE_D]) ) dihedrals[d_counter_local++ * n_frames + frame] = ddummy[0];
				d_counter++;
			}
		}
	   
	    return fail; //if anything failed return a 1, otherwise a 0
		

	}


	void modfit_dihedrals()
	{
		if (type_n_dofs[TYPE_D] == 0) return;
		const PRECISION pi=acos(-1);

	    	
		#pragma omp parallel
	    	{
	        	PRECISION modFit, binsize;
	        	int longestZeroStretch, longestZeroStretchPos, currentZeroStretch, currentZeroStretchPos;
	        	bool zeroExists;
	        	long long int histo[MODFITNBINS];
	        
	        	#pragma omp for
	        	for(int j = 0; j < type_n_dofs[TYPE_D]; j++) 
			{ //for all dihedrals
	        		//first build a histogram of the dihedral values over the trajectory
	        		for(int k=0; k<MODFITNBINS; k++) histo[k] = 0;
	        
	            		binsize = ( 2 * pi + 5e-9 * (sizeof(PRECISION) == sizeof(float) ? 100000 : 1) ) / MODFITNBINS;
	            		for(int i = 0; i < n_frames; i++) histo[ int( ( dihedrals[j * n_frames + i] ) / binsize ) ] += 1;
	            		
				zeroExists = false;
	            		for(int k=0; k<MODFITNBINS; k++) zeroExists=zeroExists||(histo[k]==0);
				
	            		if(zeroExists) 
				{ //if any of the bins of the histogram is empty find the longest consecutive stretch of  empty bins
	                		longestZeroStretch = 0;
	                		currentZeroStretch = 0;
	                		longestZeroStretchPos = -1;
	                		for(int k = 0; k < 2*MODFITNBINS; k++) 
					{ //for all bins of the histogram
	                    			int l = k % MODFITNBINS; //taking car of zero stretches which span the boundaries
	                    			if( (currentZeroStretch == 0) && (histo[l] == 0) ) 
						{ //find and save a beginning zero stretch
	                        			currentZeroStretch = 1;
	                        			currentZeroStretchPos = k;
	                    			}
	                    			if( (currentZeroStretch > 0) && (histo[l] == 0) ) 
						{
	                        			currentZeroStretch+=1;
	                    			}
	                    			if( (currentZeroStretch > 0) && ( histo[l] != 0) ) 
						{ //and the end of it. If it is currently the longest zero stretch, save it
	                        			if(currentZeroStretch > longestZeroStretch) 
							{
	                            				longestZeroStretch = currentZeroStretch;
	                            				longestZeroStretchPos = currentZeroStretchPos;
	                        			}
	                        			currentZeroStretch = 0;
	                    			}
	                		}
	            		}
	            		else 
				{ //if none of the bins is empty
	                		longestZeroStretchPos = 0;  //misuse the zeroStretch variables for determining the minimum
	                		longestZeroStretch = histo[0];
	                		for(int k = 0; k < MODFITNBINS; k++) 
					{
	                    			if(histo[k] < longestZeroStretch) 
						{
	                        			longestZeroStretch = histo[k];
	                        			longestZeroStretchPos = k;
	                    			}
	                		}
	            		}
	            		modFit = 2 * pi - (longestZeroStretchPos + 0.5) * binsize; //calculate the shift to put the zero stretch to the 2pi end
				for(int k = 0; k < n_frames; k++) 
				{							
					dihedrals[j * n_frames + k] = dihedrals[j * n_frames + k] + modFit - 2 * pi 
									* int( (dihedrals[j * n_frames + k ] + modFit ) /(2 * pi) );   //and apply it taking care of circularity
	            			

				}
	        	}
	    	}   
	}

	void calculate_extrema()
	{
		#pragma omp parallel
		{
			PRECISION tmpMin,tmpMax;

			#pragma omp for
			for(int j = 0; j < n_dofs; j++) 
			{ //for all dofs 
    				tmpMax = block_start[j * n_frames];
    				tmpMin = block_start[j * n_frames];
				//cout<<tmpMax<<endl;
    				for(int i = 1; i < n_frames; i++) 
				{ //and all frames
        				if(block_start[j * n_frames + i] > tmpMax) 
					{
            					tmpMax = block_start[j * n_frames + i];
        				}
        				if(block_start[j * n_frames + i] < tmpMin) 
					{
            					tmpMin = block_start[j * n_frames + i];   //find the maximum and minmum values
        				}
    				}
   				if( (tmpMin < 0.0) || (tmpMax < 0.0) ) 
				{
        				cerr<<"ERROR: Degree of freedom "<< dof_id_start + j <<" is smaller than 0.0"<<endl;
        				exit(EXIT_FAILURE);
    				}
    				tmpMin -= 5e-9 * ( sizeof(PRECISION) == sizeof(float) ? 1e5 :1 );//and increase the boundaries a tiny bit
    				tmpMax += 5e-9 * ( sizeof(PRECISION) == sizeof(float) ? 1e5 :1 );
    				if(tmpMin < 0.0) 
				{
        				tmpMin = 0.0;
    				}
    				if ( (tmpMax-tmpMin) < 1.0e-4) 
				{
        				tmpMax+=0.05;
    				}
    				//cout<<j<<" "<<tmpMin<<" "<<tmpMax<<endl;
				minima[dof_id_start + j] = tmpMin;//and store the calculated values
    				maxima[dof_id_start + j] = tmpMax;
			}
		}
	}
	
	void calculate_entropy1D()
	{
		#pragma omp parallel
	    	{   
			PRECISION binsize, probDens, binval, plnpsum, Jac;
	        	long long int histo[n_bins];
	        	int occupbins;
				#pragma omp for
		        	for(int j = dof_id_start; j <= dof_id_end; j++) 
				{ //for all dofs (using threads)
		            		for(int k = 0; k < n_bins; k++) 
					{
		                		histo[k] = 0;   //initialize a histogram with zeros
		            		}
		            		binsize = (maxima[j]  - minima[j]) / n_bins; //and calculate the size of the bins
		            		for(int i = 0; i < n_frames; i++) 
					{ // and fill the histogram using all frames of the trajectory
		                		histo[ int( (block_start[ (j - dof_id_start) * n_frames + i] - minima[j]) / binsize ) ] += 1;
		            		}
		            		occupbins=0; //then use the histogram to calculate the (discretized) entropy, taking care of the Jacobian
		        	    	plnpsum=0;
		        	    	binval = minima[j] + (binsize / 2.0);
		        	    	for(int k = 0; k < n_bins ; k++) 
					{
		        	        	switch( get_dof_type_from_id(j, n_dihedrals) ) 
						{
		        	            		case TYPE_B : Jac = binval*binval; break;
		        	            		case TYPE_A : Jac = sin(binval); break;
		        	            		case TYPE_D : Jac = 1; break;
		        	        	}
		        	        	probDens = histo[k] / (n_frames * binsize * Jac);
		        	        	if (probDens > 0) 
						{
		        	            		plnpsum = plnpsum + Jac * probDens * log(probDens);
		        	            		occupbins = occupbins + 1;
		        	        	}
		        	        	binval += binsize;
		        	    	}
		        	    	plnpsum = -plnpsum * binsize;
		        		result_entropy1D[j] = plnpsum + (occupbins - 1.0) / (2.0 * n_frames); //and apply Herzel entropy unbiasing
				}

	    	}   
	}
};


class GPU_RAM_Layout
{
	public:
		unsigned int dofs_per_block;
		PRECISION* dof_block_1;
		PRECISION* dof_block_2;
		PRECISION* result_fwd_1;
		PRECISION* result_fwd_2;
		PRECISION* result_all2all;
		unsigned int* occupied_bins;
		unsigned int* histograms;

		GPU_RAM_Layout(unsigned int n_frames, unsigned int n_bins, unsigned long long int gpu_n_bytes, char* gpu_ram_start)
		{
			// calculate the maximum number of dofs (for one of the two dof_blocks) so that everything still fits into GPU RAM 
			double a = (2 * n_frames - 1) * sizeof(PRECISION) - sizeof(unsigned int);
			double b = (n_bins * n_bins + 2) * sizeof(unsigned int)	+ 2 * sizeof(PRECISION);
			this->dofs_per_block = (unsigned int)( (-a/2 + sqrt(a * a / 4 + gpu_n_bytes * b) ) /b);		

			// set the pointers for partitioning the GPU RAM according to the calculated dofs_per_block
			dof_block_1 = (PRECISION*) gpu_ram_start;
			dof_block_2 = dof_block_1 + dofs_per_block * n_frames;
			result_fwd_1 = dof_block_2 + dofs_per_block * n_frames;
			result_fwd_2 = result_fwd_1 + dofs_per_block * (dofs_per_block - 1);
			result_all2all =  result_fwd_2 + dofs_per_block * (dofs_per_block - 1);
			occupied_bins = (unsigned int*) (result_all2all + dofs_per_block * dofs_per_block);
			histograms = occupied_bins + (2 * dofs_per_block - 1) * dofs_per_block;
		}
};

class CPU_RAM_Layout
{
	public:
		unsigned int dofs_per_block;
		PRECISION* dof_block_1;
		PRECISION* dof_block_2;
		PRECISION* result_entropy1D;
		PRECISION* result_entropy1D_b;
		PRECISION* result_entropy1D_a;
		PRECISION* result_entropy1D_d;
		PRECISION* result_entropy2D;
		PRECISION* result_entropy2D_bb;
		PRECISION* result_entropy2D_ba;
		PRECISION* result_entropy2D_bd;
		PRECISION* result_entropy2D_aa;
		PRECISION* result_entropy2D_ad;
		PRECISION* result_entropy2D_dd;
		PRECISION* extrema;
		PRECISION* minima;
		PRECISION* maxima;
		PRECISION* tmp_result_entropy;
		unsigned int* tmp_result_occupied_bins;
		double* tmp_read;

		CPU_RAM_Layout(unsigned int n_frames, unsigned long long int cpu_n_bytes, char* cpu_ram_start, unsigned int gpu_dofs_per_block, unsigned int n_dihedrals)
		{
        		unsigned int n_dofs_total = 3 * (n_dihedrals + 1);
			unsigned int n_bonds = n_dihedrals + 2;
			unsigned int n_angles = n_dihedrals + 1;
			// calculate the maximum number of dofs (for one of the two dof_blocks) so that everything still fits into CPU RAM
			dofs_per_block = (unsigned int)( ( cpu_n_bytes - n_dofs_total * ( ( n_dofs_total + 3 ) * sizeof(PRECISION) + sizeof(double) ) 
							+ ( 2 * gpu_dofs_per_block - 1 ) * gpu_dofs_per_block * ( sizeof(PRECISION) + sizeof(unsigned int) ) )
							/ ( 2 * n_frames * sizeof(PRECISION) ) );
			if(dofs_per_block < gpu_dofs_per_block)
			{
				cerr<<"WARNING: You probably have a GPU with a lot of RAM but your CPU RAM is rather small. ";
				cerr<<"I recommend to get more CPU RAM, as this should significantly enhance performance."<<endl;	
			}
			// if all dofs fit into RAM, still set up two blocks to be consistent with the algorithm
			if(2 * dofs_per_block >= n_dofs_total){
				dofs_per_block = n_dofs_total / 2;
				dofs_per_block += n_dofs_total % 2;
			}

			// set the pointers for partitioning the CPU RAM according to the calculated dofs_per_block
			dof_block_1 = (PRECISION*) cpu_ram_start;
			dof_block_2 = dof_block_1 + dofs_per_block * n_frames;
			
			result_entropy1D = dof_block_2 + dofs_per_block * n_frames;
			result_entropy1D_b = result_entropy1D;
			result_entropy1D_a = result_entropy1D_b + n_bonds;
			result_entropy1D_d = result_entropy1D_a + n_angles;
			
			result_entropy2D = result_entropy1D + n_dofs_total;
			result_entropy2D_bb = result_entropy2D;
			result_entropy2D_ba = result_entropy2D_bb + n_bonds * (n_bonds - 1) / 2;
			result_entropy2D_bd = result_entropy2D_ba + n_bonds * n_angles;
			result_entropy2D_aa = result_entropy2D_bd + n_bonds * n_dihedrals;
			result_entropy2D_ad = result_entropy2D_aa + n_angles * (n_angles - 1) / 2;
			result_entropy2D_dd = result_entropy2D_ad + n_angles * n_dihedrals;
			
            
			extrema = result_entropy2D + n_dofs_total * (n_dofs_total - 1) / 2;
			minima = extrema;
			maxima = extrema + n_dofs_total;
			tmp_result_entropy = maxima + n_dofs_total;
			tmp_result_occupied_bins = (unsigned int*)(tmp_result_entropy + (2 * gpu_dofs_per_block - 1) * gpu_dofs_per_block);
			tmp_read = (double*)(tmp_result_occupied_bins + (2 * gpu_dofs_per_block - 1) * gpu_dofs_per_block);
		}

};



class RAM{
	public:
		char *cpu_ram_start;
                char *cpu_ram_end;
                unsigned long long int cpu_n_bytes;
		char *gpu_ram_start;
                char *gpu_ram_end;
                unsigned long long int gpu_n_bytes;
		GPU_RAM_Layout* gpu_ram_layout;
		CPU_RAM_Layout* cpu_ram_layout;
		unsigned int n_dihedrals;
		unsigned int n_dofs_total;
		vector<CPU_RAM_Block> blocks;

		RAM(unsigned long long int cpu_n_bytes, unsigned long long int gpu_n_bytes, unsigned int n_dihedrals, unsigned int n_frames, 
			unsigned int n_bins, ifstream* bat_file, streamoff file_dofs_begin, unsigned char precision_traj)
		{
			cpu_ram_start = new char [cpu_n_bytes];
			cpu_ram_end = cpu_ram_start + cpu_n_bytes - 1;
                        this->cpu_n_bytes = cpu_n_bytes; 
			gpuErrchk( hipMalloc((void**) &gpu_ram_start, gpu_n_bytes) );
                        gpu_ram_end = gpu_ram_start + gpu_n_bytes - 1;
                        this->gpu_n_bytes = gpu_n_bytes;
			gpu_ram_layout = new GPU_RAM_Layout(n_frames, n_bins, gpu_n_bytes, gpu_ram_start);
			this->n_dihedrals = n_dihedrals;
            this->n_dofs_total = 3 * (n_dihedrals + 1);
			cpu_ram_layout = new CPU_RAM_Layout(n_frames, cpu_n_bytes, cpu_ram_start, gpu_ram_layout->dofs_per_block, n_dihedrals);
			for(unsigned int i = 0; i < n_dofs_total; i+=cpu_ram_layout->dofs_per_block)
			{
				unsigned int end = i + cpu_ram_layout->dofs_per_block - 1;
				if(end > n_dofs_total - 1) end = n_dofs_total - 1;
				blocks.push_back(*new CPU_RAM_Block(i, end, gpu_ram_layout->dofs_per_block, n_dihedrals, n_frames, bat_file, file_dofs_begin, precision_traj, 
									cpu_ram_layout->minima, cpu_ram_layout->maxima, n_bins, cpu_ram_layout->result_entropy1D));
			} 

			
		}


};


#include <algorithm>
char* getCmdOption(char ** begin, char ** end, const string & option)
{
    char ** itr = find(begin, end, option);
    if (itr != end && ++itr != end)
    {
        return *itr;
    }
    return 0;
}

bool cmdOptionExists(char** begin, char** end, const string& option)
{
    return find(begin, end, option) != end;
}


int main(int argc, char *argv[]){

	//start the stopwatch for the execution time
	timeval tv_start,tv_end;
	gettimeofday (&tv_start, NULL);
	
	int deviceCount;    
	gpuErrchk(hipGetDeviceCount(&deviceCount));
	
	
	unsigned int device=0;//TODO: implement choices for graphics card
	cout<<"Found "<<deviceCount<<" CUDA device(s). Chose CUDA device number "<<device<<"."<<endl;
	struct hipDeviceProp_t prop;
	gpuErrchk(hipGetDeviceProperties(&prop,device)); 	
	cout<<"Device name: "<<prop.name<<endl;
	cout<<"CUDA capability: "<<prop.major<<"."<<prop.minor<<endl;
	cout<<"Global memory: "<<prop.totalGlobalMem/1024/1024<<" MiB"<<endl;
	cout<<"Shared memory per block: "<<prop.sharedMemPerBlock/1024<<" kiB"<<endl;
	cout<<"Maximum threads per block dimension: "<<prop.maxThreadsDim[0]<<" "<<prop.maxThreadsDim[1]<<" "<<prop.maxThreadsDim[2]<<endl;
	cout<<"Maximum blocks per grid dimension: "<<prop.maxGridSize[0]<<" "<<prop.maxGridSize[1]<<" "<<prop.maxGridSize[2]<<endl;
	cout<<"Warp size: "<<prop.warpSize<<endl;
	cout<<endl<<endl;    
	
	//int threads_per_block = prop.warpSize*WARPMULTIPLES; 
	
	int precision_traj, n_frames;
	unsigned int n_bins;
	vector< vector <int> > dihedrals_top;
	vector <float> masses;
	vector <string> residues;
	vector <int> residueNumbers;
	vector <string> atomNames;
	vector <string> belongsToMolecule;
	
	if(argc!=7) 
	{
		cerr<<"USAGE: "<<argv[0]<<" -f input.bat -o entropy.par -b #bins\n";
	    	exit(EXIT_FAILURE);	
	}
	bool fail = false;
	fail |=  !cmdOptionExists(argv, argv+argc, "-f");
	fail |=  !cmdOptionExists(argv, argv+argc, "-o");
	fail |=  !cmdOptionExists(argv, argv+argc, "-b");
		          
	if(fail)
	{
		//check for correct command line options
		cerr<<"USAGE: "<<argv[0]<<" -f input.bat -o entropy.par -b #bins\n";
		exit(EXIT_FAILURE);
	}
	                
	string tmp1(getCmdOption(argv, argv+argc, "-f")); //first argument is the .bat trajectory file
	string tmp2(getCmdOption(argv, argv+argc, "-o")); //second argument is the .par output file
	char *ptr,*type1,*type2;
	char delimiter[] = ".";
	
	ptr = strtok((char*)tmp1.c_str(), delimiter);
	while(ptr != NULL) 
	{
		type1 = ptr;
		ptr = strtok(NULL, delimiter);
	}
	
	ptr = strtok((char*)tmp2.c_str(), delimiter);
	while(ptr != NULL) 
	{
		type2=ptr;
		ptr = strtok(NULL, delimiter);
	}
	if((strcmp(type1,"bat"))||(strcmp(type2,"par"))) 
	{
		//check for the extensions of the input and output file
		cerr<<"USAGE: "<<argv[0]<<" -f input.bat -o entropy.par -b #bins\n";
		exit(EXIT_FAILURE);
	}
	if(sscanf(getCmdOption(argv, argv+argc, "-b"),"%ud",&n_bins)!=1) 
	{
		 //read the number of bins and check for correctness
		cerr<<"ERROR: Could not read number of bins from command line! Aborting"<<endl;   		
		exit(EXIT_FAILURE);
	}
	
		
	ifstream bat_file;
	ofstream par_file;
	cout<<"Reading file "<<getCmdOption(argv, argv+argc, "-f")<<" .\n";
	//open the input/output files
	bat_file.open(getCmdOption(argv, argv+argc, "-f"), ios::binary | ios::in );
	par_file.open(getCmdOption(argv, argv+argc, "-o"),ios::binary | ios::out);
	if(!(bat_file.is_open())) 
	{
	    cerr<<"ERROR: Could not open file "<<getCmdOption(argv, argv+argc, "-f")<<" ! Aborting."<<endl;
	    exit(EXIT_FAILURE);
	}
	if(!(par_file.is_open())) 
	{
	    cerr<<"ERROR: Could not open file "<<getCmdOption(argv, argv+argc, "-f")<<" ! Aborting."<<endl;
	    exit(EXIT_FAILURE);
	}
	
	//and read the header of the trajectory
	if(read_BAT_header(&bat_file, &precision_traj, &n_frames, &dihedrals_top, &masses, &residues, &residueNumbers, &atomNames, &belongsToMolecule)!=0) 
	{
	    cerr<<"AN ERROR HAS OCCURED WHILE READING THE HEADER OF THE FILE " <<getCmdOption(argv, argv+argc, "-b")<<" . QUITTING PROGRAM.\n";
	    exit(EXIT_FAILURE);
	}
	unsigned int n_dihedrals=dihedrals_top.size();
	cout<<getCmdOption(argv, argv+argc, "-b")<<" specs:"<<endl;
	cout<<"Precision: "<<(precision_traj == 1 ? "double" : "single")<<" #Atoms: "<<n_dihedrals + 3<<" #Frames: "<<n_frames<<endl;  // ---------------------------------------------------
	streamoff file_dofs_begin = bat_file.tellg();
	
	//and write the header of the output (.par) file
	if(write_PAR_header(&par_file,n_dihedrals,precision_traj,n_frames,&dihedrals_top, &masses, 
				n_bins, n_bins, n_bins, n_bins, n_bins, n_bins, &residues, &residueNumbers, &atomNames, &belongsToMolecule) != 0) 
	{
	    cerr<<"AN ERROR HAS OCCURED WHILE WRITING THE HEADER OF THE FILE " <<getCmdOption(argv, argv+argc, "-o")<<" . QUITTING PROGRAM.\n";
	    exit(EXIT_FAILURE);
	}










	unsigned long long int cpu_ram_available = static_cast<unsigned long long int>(1024)*1024*1024*4;
	unsigned long long int gpu_ram_available = static_cast<unsigned long long int>(1024)*1024*1024*1;


	RAM ram(cpu_ram_available, gpu_ram_available, n_dihedrals, n_frames, n_bins, &bat_file, file_dofs_begin, precision_traj);
	//ram.cpu_ram_layout->result_entropy1D[0] = 42;
	//ram.cpu_ram_layout->result_entropy1D[-1] = 42;

	//cout<<ram.gpu_ram_layout->dofs_per_block<<" "<<ram.cpu_ram_layout->dofs_per_block<<endl;

	//ram.cpu_ram_layout->tmp_result_occupied_bins[0] = 1;


	for (unsigned int i = 0; i < ram.blocks.size() - 1; i++)
	{	cout<<"Deploying Block "<<i+1<<" to RAM bank 1."<<endl;
		ram.blocks[i].deploy(ram.cpu_ram_layout->dof_block_1);
		//cout<<ram.cpu_ram_layout->result_entropy1D[-1]<<" "<<ram.cpu_ram_layout->result_entropy1D[0]<<endl<<endl;
		for (unsigned int j = i + 1; j < ram.blocks.size(); j++)
		{
			cout<<ram.blocks[i].dof_id_start<<" "<<ram.blocks[i].dof_id_end<<" "<<ram.blocks[j].dof_id_start<<" "<<ram.blocks[j].dof_id_end<<endl;
			cout<<"Deploy Block "<<j+1<<" to RAM bank 2."<<endl;
			ram.blocks[j].deploy(ram.cpu_ram_layout->dof_block_2);
			//cout<<ram.cpu_ram_layout->result_entropy1D[-1]<<" "<<ram.cpu_ram_layout->result_entropy1D[0]<<endl<<endl;

		}
	}

	/*for(int i = 0; i<3*n_dihedrals + 3; i++)
	{
		ram.cpu_ram_layout->minima[i] = 0;
		ram.cpu_ram_layout->maxima[i] = 0;
	}
	ram.blocks[1].deploy(ram.cpu_ram_layout->dof_block_1);*/


	//for(int i = 0; i<3*n_dihedrals + 3; i++) cout<<i<<" "<<ram.cpu_ram_layout->minima[i]<<" "<<ram.cpu_ram_layout->maxima[i]<<endl;
	//timings are written to stdout
    
    //write out the results to the binary .par file and measure time
    
    if(write_PAR_body(&par_file, n_dihedrals, ram.cpu_ram_layout->result_entropy1D_b, ram.cpu_ram_layout->result_entropy1D_a, ram.cpu_ram_layout->result_entropy1D_d, 
                        ram.cpu_ram_layout->result_entropy2D_bb, ram.cpu_ram_layout->result_entropy2D_ba, ram.cpu_ram_layout->result_entropy2D_bd, 
                        ram.cpu_ram_layout->result_entropy2D_aa, ram.cpu_ram_layout->result_entropy2D_ad, ram.cpu_ram_layout->result_entropy2D_dd) !=0 ) {
        cerr<<"AN ERROR HAS OCCURED WHILE WRITING THE FILE " <<getCmdOption(argv, argv+argc, "-o")<<" .\n";
        exit(EXIT_FAILURE);
    }
    


    //timings are written out
    par_file.close();
    
    
	par_file.close();
	gettimeofday (&tv_end, NULL);
	cout<<endl<<endl;
	cout<<"Total execution time: "<<tv_end.tv_sec+1e-6 * tv_end.tv_usec-tv_start.tv_sec-1e-6 * tv_start.tv_usec<<endl;
 	cout<<"PROGRAM FINISHED SUCCESSFULLY."<<endl<<endl<<endl;
	
	hipDeviceReset();
	return 0;
}



